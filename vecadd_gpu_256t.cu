
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  int index = threadIdx.x;  //index of the current thread within its block
  int stride = blockDim.x;  //number of threads in the block
  for (int i = index; i < n; i += stride) //stride through the array with parallel threads.
      y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1<<26; // 64M elements
  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

 // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }
  add<<<1, 256>>>(N, x, y); //changed thread size

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;
  
 // Free memory
  hipFree(x);
  hipFree(y);

  return 0;
}